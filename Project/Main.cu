﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define N (1<<25)
#define B 4096

typedef unsigned int uint32;

__global__ void gpuHistogram(uint32* bufferIn, uint32 bufferInSize, uint32* bufferOut, uint32 bufferOutSize)
{
	uint32 tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < bufferInSize) atomicAdd(&bufferOut[bufferIn[tid]], 1);
}

__global__ void gpuHistogram2(uint32* bufferIn, uint32 bufferInSize, uint32* bufferOut, uint32 bufferOutSize)
{
	__shared__ uint32 groupshared[B];

	uint32 tid = threadIdx.x;
	uint32 gid = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = 0; i < B; i += blockDim.x)
	{
		groupshared[i + tid] = 0;
	}

	if (gid < bufferInSize)
	{
		atomicAdd(&groupshared[bufferIn[gid]], 1);
	}

	for (int i = 0; i < B; i += blockDim.x)
	{
		atomicAdd(&bufferOut[i + tid], groupshared[i + tid]);
	}
}

__global__ void gpuSaturate(uint32* buffer, uint32 bufferSize, uint32 valueMin, uint32 valueMax)
{
	uint32 tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < bufferSize) buffer[tid] = buffer[tid] > valueMax ? valueMax : buffer[tid];
}

int main()
{
	uint32* cpuBuffer1;
	uint32* cpuBuffer2;
	uint32* gpuBuffer1;
	uint32* gpuBuffer2;

	uint32 threads = 0;
	uint32 blocks  = 0;
	uint32 errors  = 0;

	srand(time(NULL));

	hipHostAlloc((void**)&cpuBuffer1, N * sizeof(uint32), hipHostMallocDefault);
	hipHostAlloc((void**)&cpuBuffer2, B * sizeof(uint32), hipHostMallocDefault);

	hipMalloc((void**)&gpuBuffer1, N * sizeof(uint32));
	hipMalloc((void**)&gpuBuffer2, B * sizeof(uint32));

	for (int i = 0; i != N; ++i)
	{
		cpuBuffer1[i] = (uint32)rand() % (uint32)B;
	}

	hipMemcpy(gpuBuffer1, cpuBuffer1, N * sizeof(uint32), hipMemcpyHostToDevice);
	hipMemset(gpuBuffer2, 0, B * sizeof(uint32));

	threads = 1024;
	blocks  = (N + threads - 1) / threads;
	gpuHistogram<<<blocks, threads>>>(gpuBuffer1, N, gpuBuffer2, B);

	threads = 64;
	blocks  = (B + threads - 1) / threads;
	gpuSaturate<<<blocks, threads>>>(gpuBuffer2, B, 0, 127);

	hipDeviceSynchronize();
	hipMemcpy(cpuBuffer2, gpuBuffer2, B * sizeof(uint32), hipMemcpyDeviceToHost);

	uint32 values[B];
	memset(values, 0, B * sizeof(uint32));

	for (int i = 0; i != N; ++i)
	{
		values[cpuBuffer1[i]] += values[cpuBuffer1[i]] < 127 ? 1 : 0;
	}

	for (int i = 0; i != B; ++i)
	{
		errors += (cpuBuffer2[i] != values[i]);
	}

	printf("Elements: %u\n", N);
	printf("Errors:   %u\n", errors);

	hipFree(gpuBuffer1);
	hipFree(gpuBuffer2);

	hipHostFree(cpuBuffer1);
	hipHostFree(cpuBuffer2);

	return 0;
}